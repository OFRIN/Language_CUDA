#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include ""

using namespace std;

#define N (2048 * 2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

void random_ints(int* a, int n)
{
	for (int i = 0; i < n; ++i)
		a[i] = rand();
}

int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	// Allocate space for device copies of a, b, c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	// Setup input values
	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	random_ints(a, N);
	random_ints(b, N);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add << <N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	cout << "# a" << endl;
	for (int i = 0; i<10; i++)
		cout << a[i] << " ";
	cout << endl;

	cout << "# b" << endl;
	for (int i = 0; i<10; i++)
		cout << b[i] << " ";
	cout << endl;

	cout << "# c" << endl;
	for (int i = 0; i<10; i++)
		cout << c[i] << " ";
	cout << endl;

	// Cleanup
	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
