#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <>

using namespace std;

#define THREAD_N 512

__global__ void add(int *a, int *b, int *c) {
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random_ints(int* a, int N)
{
	for (int i = 0; i < N; ++i)
		a[i] = rand();
}

int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = THREAD_N * sizeof(int);

	// Allocate space for device copies of a, b, c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	// Setup input values
	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	random_ints(a, THREAD_N);
	random_ints(b, THREAD_N);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU with N threads
	add << <1, THREAD_N >> >(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	cout << "# a" << endl;
	for (int i = 0; i<10; i++)
		cout << a[i] << " ";
	cout << endl;

	cout << "# b" << endl;
	for (int i = 0; i<10; i++)
		cout << b[i] << " ";
	cout << endl;

	cout << "# c" << endl;
	for (int i = 0; i<10; i++)
		cout << c[i] << " ";
	cout << endl;

	// Cleanup
	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
